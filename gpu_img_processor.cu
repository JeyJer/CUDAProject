#include <opencv2/opencv.hpp>
#include <vector>
#include <cstring>
#include <gpu/gpu_img_transform_stream.cuh>

#include "gpu/gpu_utilities.cuh"
#include "common/utilities.hpp"
#include "gpu/gpu_img_transform.cuh"

int main(int argc, char **argv)
{
    std::string img_out;  // default values in utilities.hpp
    std::string img_in;

    GpuUtilMenuSelection menuSelection;
    GpuUtilMenuSelection::initParameters(img_in, img_out, menuSelection, argc, argv);

    GpuUtilExecutionInfo info;

    cv::Mat m_in = cv::imread(img_in, cv::IMREAD_UNCHANGED);

    unsigned char* rgb_out = nullptr, *rgb_in = nullptr;
    hipHostMalloc(&rgb_in, 3 * m_in.rows * m_in.cols);
    hipHostMalloc(&rgb_out, 3 * m_in.rows * m_in.cols);

    memcpy(rgb_in, m_in.data, 3 * m_in.rows * m_in.cols);

    cv::Mat m_in_aux(m_in.rows, m_in.cols, CV_8UC3, rgb_in);
    cv::Mat m_out(m_in.rows, m_in.cols, CV_8UC3, rgb_out);

    int (*fnc_exec) (cv::Mat&, cv::Mat&, GpuUtilExecutionInfo& );

    if( menuSelection.nb_stream == 0) {
        if (!menuSelection.use_shared)
            fnc_exec = GpuImgTransform::execute;
        else
            fnc_exec = GpuImgTransform::executeSharedMemMode;
    }else {
        if( menuSelection.nb_stream > m_in.rows ) menuSelection.nb_stream = m_in.rows;

        while(m_in.rows % menuSelection.nb_stream != 0) menuSelection.nb_stream--;

        if (!menuSelection.use_shared)
            fnc_exec = GpuImgTransformStream::execute;
        else
            fnc_exec = GpuImgTransformStream::executeSharedMemMode;
    }
    for( int i = 0; i < menuSelection.enabled_filters.size(); i++){
        if( i > 0 ) swapPointers(&m_in_aux.data, &m_out.data);

        EffectStyle filter = menuSelection.enabled_filters.at(i);
        set_convolution_properties(info.conv_properties, filter);
        int conv_mat_length = info.conv_properties.size * info.conv_properties.size;

        char conv_mat[conv_mat_length];

        info.conv_matrix = conv_mat;
        info.nb_pass = menuSelection.nb_pass.at(i);
        info.nb_streams = menuSelection.nb_stream;
        info.block.x = menuSelection.block.dimX;
        info.block.y = menuSelection.block.dimY;

        copyReverse(conv_mat, filter, conv_mat_length);

        (*fnc_exec)(m_in_aux, m_out, info );

    }

    cv::imwrite(img_out, m_out);

    hipHostFree(rgb_in);
    hipHostFree(rgb_out);

    return 0;
}
