#include <opencv2/opencv.hpp>
#include <vector>
#include <cstring>
#include <gpu/gpu_img_transform_stream.cuh>

#include "common/menu_lib.hpp"
#include "common/utilities.hpp"
#include "gpu/gpu_img_transform.cuh"

int main(int argc, char **argv)
{
    std::string img_out = "/mnt/data/tsky-19/eclipsec/CUDAProjectV2/out.jpg";
    std::string img_in = "/mnt/data/tsky-19/eclipsec/CUDAProjectV2/in.jpg";

    MenuSelection menuSelection;
    initParameters(img_in, img_out, menuSelection, argc, argv);

    ExecutionInfo info;

    cv::Mat m_in = cv::imread(img_in, cv::IMREAD_UNCHANGED);

    //std::vector< unsigned char > g( rows * cols );
    // Allocation de l'image de sortie en RAM côté CPU.
    auto rows = m_in.rows;
    auto cols = m_in.cols;
    unsigned char* rgb_in_aux = nullptr;
    hipHostMalloc(&rgb_in_aux, 3 * m_in.rows * m_in.cols);
    cv::Mat m_out(m_in.rows, m_in.cols, CV_8UC3, rgb_in_aux);

    int (*fnc_exec) (cv::Mat&, cv::Mat&, ExecutionInfo& );

    // menuSelection.nb_stream = 20;
    if( menuSelection.nb_stream == 0) {
        if (!menuSelection.use_shared)
            fnc_exec = GpuImgTransform::execute;
        else
            fnc_exec = GpuImgTransform::executeSharedMemMode;
    }else {
        if (!menuSelection.use_shared)
            fnc_exec = GpuImgTransformStream::execute;
        else
            fnc_exec = GpuImgTransformStream::executeSharedMemMode;
    }
    for( int i = 0; i < menuSelection.enabled_filters.size(); i++){
        EffectStyle filter = menuSelection.enabled_filters.at(i);
        set_convolution_properties(info.conv_properties, filter);
        int conv_mat_length = info.conv_properties.size * info.conv_properties.size;

        char conv_mat[conv_mat_length];
        info.set(conv_mat, menuSelection.nb_pass.at(i), menuSelection.block.dimX,
                 menuSelection.block.dimY, menuSelection.nb_stream);

        copyReverse(conv_mat, filter, conv_mat_length);

        (*fnc_exec)(m_in, m_out, info );

        memcpy(m_in.data, m_out.data, 3 * rows * cols  * sizeof(unsigned char));

    }

    cv::imwrite(img_out, m_out);

    hipHostFree(rgb_in_aux);

    return 0;
}
