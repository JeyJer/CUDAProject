#include <opencv2/opencv.hpp>
#include <vector>
#include <cstring>
#include <gpu/gpu_img_transform_stream.cuh>

#include "gpu/gpu_utilities.cuh"
#include "common/utilities.hpp"
#include "gpu/gpu_img_transform.cuh"

int main(int argc, char **argv)
{
    std::string img_out;  // default values in utilities.hpp
    std::string img_in;

    GpuUtilMenuSelection menuSelection;
    GpuUtilMenuSelection::initParameters(img_in, img_out, menuSelection, argc, argv);

    GpuUtilExecutionInfo info;

    cv::Mat m_in = cv::imread(img_in, cv::IMREAD_UNCHANGED);

    unsigned char* rgb_in_aux = nullptr;
    hipHostMalloc(&rgb_in_aux, 3 * m_in.rows * m_in.cols);
    cv::Mat m_out(m_in.rows, m_in.cols, CV_8UC3, rgb_in_aux);

    int (*fnc_exec) (cv::Mat&, cv::Mat&, GpuUtilExecutionInfo& );

    menuSelection.nb_stream = 60;   // DEBUG: pour les testes
    if( menuSelection.nb_stream == 0) {
        if (!menuSelection.use_shared)
            fnc_exec = GpuImgTransform::execute;
        else
            fnc_exec = GpuImgTransform::executeSharedMemMode;
    }else {
        if (!menuSelection.use_shared)
            fnc_exec = GpuImgTransformStream::execute;
        else
            fnc_exec = GpuImgTransformStream::executeSharedMemMode;
    }
    for( int i = 0; i < menuSelection.enabled_filters.size(); i++){
        EffectStyle filter = menuSelection.enabled_filters.at(i);
        set_convolution_properties(info.conv_properties, filter);
        int conv_mat_length = info.conv_properties.size * info.conv_properties.size;

        char conv_mat[conv_mat_length];

        info.conv_matrix = conv_mat;
        info.nb_pass = menuSelection.nb_pass.at(i);
        info.nb_streams = menuSelection.nb_stream;
        info.block.x = menuSelection.block.dimX;
        info.block.y = menuSelection.block.dimY;

        copyReverse(conv_mat, filter, conv_mat_length);

        info.nb_pass = 20;  // DEBUG: pour les testes
        (*fnc_exec)(m_in, m_out, info );

    }

    cv::imwrite(img_out, m_out);

    hipHostFree(rgb_in_aux);

    return 0;
}
