#include "hip/hip_runtime.h"
#include "gpu/gpu_img_transform_stream.cuh"

StreamInfo::StreamInfo(double size, long rows): size(size), rows(rows){
    size_effective = size;
}
StreamInfo::StreamInfo(const StreamInfo &streams_info, int nb_streams, long cols, ConvolutionMatrixProperties &conv_prop){
    if( nb_streams == 1){
        size_effective = streams_info.size;
        size = streams_info.size;
        rows = streams_info.rows;
        size_effective = size;
    } else {
        // add k rows per stream
        rows = streams_info.rows / nb_streams - 2 * conv_prop.start_index;
        size = streams_info.size / nb_streams - 2 * (double)conv_prop.start_index * (double)cols * 3;

        // effective output size
        size_effective = streams_info.size / nb_streams;
    }
}

__global__ void transform_img_stream(unsigned char* input, unsigned char* output, std::size_t nb_cols, std::size_t nb_rows,
                              char * conv_mat, ConvolutionMatrixProperties *conv_mat_properties)
{
    long ith_col = blockIdx.x * blockDim.x + threadIdx.x;
    long jth_row = blockIdx.y * blockDim.y + threadIdx.y;

    if (ith_col + conv_mat_properties->start_index >= 0 && ith_col < nb_cols + conv_mat_properties->start_index &&
        jth_row + conv_mat_properties->start_index >= 0 && jth_row < nb_rows + conv_mat_properties->start_index)
    {
        long j_local = jth_row + conv_mat_properties->start_index;
        long i_local;

        long i_max = ith_col + conv_mat_properties->start_index + conv_mat_properties->size;
        long j_max = j_local + conv_mat_properties->size;
        long rgb[3] = {0, 0, 0};
        for( int j_inc = 0; j_local < j_max; j_local++){

            i_local = i_max - conv_mat_properties->size;
            long index =  3 * (j_local * (long)nb_cols + i_local);
            for(  ; i_local < i_max; i_local++, j_inc++ ){
                rgb[0] += conv_mat[j_inc] * input[ index ];
                rgb[1] += conv_mat[j_inc] * input[ index + 1 ];
                rgb[2] += conv_mat[j_inc] * input[ index + 2 ];

                index += 3;
            }
        }
        long j_desti = jth_row + conv_mat_properties->start_index;

        for( long i = 0, j = 3 * (j_desti * (long)nb_cols + ith_col); i < 3; i++, j++)
            output[j] = rgb[i] / conv_mat_properties->divisor;
    }
}

__global__ void transform_img_stream_shared(unsigned char* input, unsigned char* output,
                                     std::size_t nb_cols_global, std::size_t nb_rows_global,
                                     char * conv_mat, ConvolutionMatrixProperties *conv_prop)
{
    extern __shared__ unsigned char sh[];

    long ith_col_global = blockIdx.x * (blockDim.x + conv_prop->start_index - 1) + threadIdx.x;
    long jth_row_global = blockIdx.y * (blockDim.y + conv_prop->start_index - 1) + threadIdx.y;

    long ith_col = threadIdx.x;
    long jth_row = threadIdx.y;

    long nb_rows = blockDim.y;
    long nb_cols = blockDim.x;

    if (ith_col_global < nb_cols_global && jth_row_global < nb_rows_global)
    {
        long index = 3 * (jth_row * nb_cols + ith_col);
        long index_global = 3 * (jth_row_global * (long)nb_cols_global + (long)ith_col_global);
        sh[index] = input[index_global ];
        sh[index + 1] = input[index_global + 1];
        sh[index + 2] = input[index_global + 2];
    }

    __syncthreads();

    if ( ( ith_col_global > 0 && ith_col_global < nb_cols_global ) &&
         ( jth_row_global > 0 && jth_row_global < nb_rows_global) &&
         ( ith_col + conv_prop->start_index >= 0 && ith_col < nb_cols + conv_prop->start_index ) &&
         ( jth_row + conv_prop->start_index >= 0 && jth_row < nb_rows + conv_prop->start_index) )
    {
        long j_local = jth_row + conv_prop->start_index;
        long i_local;

        long i_max = ith_col + conv_prop->start_index + conv_prop->size;
        long j_max = j_local + conv_prop->size;
        long rgb[3] = {0, 0, 0};
        for( long j_inc = 0; j_local < j_max; j_local++){

            i_local = i_max - conv_prop->size;
            long index =  3 * (j_local * nb_cols + i_local);
            for(  ; i_local < i_max; i_local++, j_inc++ ){
                rgb[0] += conv_mat[j_inc] * sh[ index ];
                rgb[1] += conv_mat[j_inc] * sh[ index + 1 ];
                rgb[2] += conv_mat[j_inc] * sh[ index + 2 ];

                index += 3;
            }
        }
        long j_desti = jth_row_global + conv_prop->start_index;

        for( long i = 0, j = 3 * (j_desti * (long)nb_cols_global + ith_col_global); i < 3; i++, j++)
            output[j] = rgb[i] / conv_prop->divisor;
    }

}

void GpuImgTransformStream::initMemory(cv::Mat &m_in, ConvMatrixPointers &dev_convolution,
        Pointers &host, long size, int conv_mat_length){
    hipHostMalloc(&host.rgb.in, size);
    std::memcpy(host.rgb.in, m_in.data, size);

    hipMalloc(&dev_convolution.matrix, conv_mat_length * sizeof(char));
    hipMalloc(&dev_convolution.prop, sizeof(ConvolutionMatrixProperties));

    hipMemcpy(dev_convolution.matrix, host.convolution.matrix, conv_mat_length, hipMemcpyHostToDevice);
    hipMemcpy(dev_convolution.prop , host.convolution.prop, sizeof(ConvolutionMatrixProperties),
            hipMemcpyHostToDevice);

}

void GpuImgTransformStream::initStreamAndDevMem(StreamInfo &per_stream_info, int nb_streams, hipStream_t *streams,
        RgbPointers *dev_rgbs, unsigned char *host_rgb_in){

    long size_bytes = (long)per_stream_info.size * (long)sizeof(unsigned char);
    for( int i = 0 ; i < nb_streams; i++ ) {
        hipMalloc(&dev_rgbs[i].in, size_bytes);
        hipMalloc(&dev_rgbs[i].out, size_bytes);
    }

    for( int i = 0 ; i < nb_streams; i++ ){
        hipMemcpyAsync( dev_rgbs[i].in,host_rgb_in + (int)(i * per_stream_info.size_effective),
                         size_bytes,
                         hipMemcpyHostToDevice, streams[i] );
    }
}
void GpuImgTransformStream::swapStreamMem(StreamInfo &per_stream_info, int nb_streams, RgbPointers *dev_rgbs){
    for( int i = 0 ; i < nb_streams; i++ )  swapPointers(&dev_rgbs[i].in, &dev_rgbs[i].out);

    long size_copy_bytes = (long)(per_stream_info.size - per_stream_info.size_effective) * (long)sizeof(unsigned char);

    for( long i = 0; i < nb_streams - 1; i++){
        /*
        hipMemcpyAsync(dev_rgbs[i].in + (long)per_stream_info.size_effective, dev_rgbs[i+1].in,
                        size_copy_bytes, hipMemcpyDeviceToDevice);
        */
        hipMemcpyAsync(dev_rgbs[i].in + (long)per_stream_info.size_effective, dev_rgbs[i+1].in,
                        size_copy_bytes, hipMemcpyDeviceToDevice);
    }


}
void GpuImgTransformStream::freeMemory(RgbPointers *dev_rgbs, ConvMatrixPointers &dev_convolution, Pointers &host, int nb_streams){
    for( long i = 0 ; i < nb_streams; i++ ){
        hipFree(dev_rgbs[i].in);
        hipFree(dev_rgbs[i].in);
    }
    hipFree(dev_convolution.matrix);
    hipFree(dev_convolution.prop);

    hipHostFree(host.rgb.in);
}

int GpuImgTransformStream::execute(cv::Mat &m_in, cv::Mat &m_out, GpuUtilExecutionInfo &info)
{
    auto rows = m_in.rows;
    auto cols = m_in.cols;

    // Pointers dev;
    Pointers host;
    ConvMatrixPointers dev_convolution;
    RgbPointers dev_rgbs[ info.nb_streams ];

    hipStream_t streams[ info.nb_streams  ];

    host.convolution.prop = &info.conv_properties;
    host.convolution.matrix = info.conv_matrix;

    long i = 3 * rows * cols;
    StreamInfo streams_info((double)i,  m_in.rows);

    StreamInfo per_stream_info(streams_info, info.nb_streams, cols, info.conv_properties);

    int conv_mat_length = info.conv_properties.size * info.conv_properties.size;

    host.convolution.prop = &info.conv_properties;
    host.convolution.matrix = info.conv_matrix;

    GpuImgTransformStream::initMemory(m_in, dev_convolution, host, i,  conv_mat_length);


    for( i = 0 ; i < info.nb_streams ; i++ ) hipStreamCreate( &streams[ i ] );

    GpuImgTransformStream::initStreamAndDevMem(per_stream_info, info.nb_streams, streams, dev_rgbs, host.rgb.in);

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Mesure du temps de calcul du kernel uniquement.
    hipEventRecord(start);

    dim3 grid0((cols - 1) / (info.block.x - 1 + info.conv_properties.start_index) + 1,
               (rows - 1) / (info.block.y - 1 + info.conv_properties.start_index) + 1);


    for( i = 0 ; i < info.nb_streams ; i++ )
    {
        transform_img_stream<<< grid0, info.block, 0, streams[i] >>>(
                dev_rgbs[i].in, dev_rgbs[i].out,
                cols, per_stream_info.rows , dev_convolution.matrix, dev_convolution.prop);
    }


    for( int kth_pass = 1; kth_pass < info.nb_pass; kth_pass++){
        GpuImgTransformStream::swapStreamMem(per_stream_info, info.nb_streams, dev_rgbs);
        for( i = 0 ; i < info.nb_streams ; i++ )
        {
            transform_img_stream<<< grid0, info.block, 0, streams[i] >>>(
                    dev_rgbs[i].in, dev_rgbs[i].out,
                    cols, per_stream_info.rows , dev_convolution.matrix, dev_convolution.prop);
        }
    }

    hipDeviceSynchronize();
    hipEventRecord(stop);

    for(  i = 0 ; i < info.nb_streams ; i++ )
    {
        unsigned char *desti = m_out.data + (long)(i * (long)per_stream_info.size_effective);

        hipMemcpyAsync( desti, dev_rgbs[i].out,(long)per_stream_info.size_effective * sizeof(unsigned char),
                hipMemcpyDeviceToHost, streams[i] );
    }

    hipEventSynchronize(stop);

    for( i = 0 ; i < info.nb_streams ; i++ ) hipStreamDestroy( streams[i] );

    float duration;
    hipEventElapsedTime(&duration, start, stop);
    std::cout << "time=" << duration << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    GpuImgTransformStream::freeMemory(dev_rgbs, dev_convolution, host, info.nb_streams);
    return 0;
}

// TODO : debug [block image de dessous semble manquer <<Le dernier stream semble ne rien ecrire>> ]
int GpuImgTransformStream::executeSharedMemMode(cv::Mat &m_in, cv::Mat &m_out, GpuUtilExecutionInfo &info)
{
    auto rows = m_in.rows;
    auto cols = m_in.cols;

    // Pointers dev;
    Pointers host;
    ConvMatrixPointers dev_convolution;
    RgbPointers dev_rgbs[ info.nb_streams ];

    hipStream_t streams[ info.nb_streams  ];

    host.convolution.prop = &info.conv_properties;
    host.convolution.matrix = info.conv_matrix;

    long i = 3 * rows * cols;
    StreamInfo streams_info((double)i,  m_in.rows);

    StreamInfo per_stream_info(streams_info, info.nb_streams, cols, info.conv_properties);

    int conv_mat_length = info.conv_properties.size * info.conv_properties.size;

    host.convolution.prop = &info.conv_properties;
    host.convolution.matrix = info.conv_matrix;

    GpuImgTransformStream::initMemory(m_in, dev_convolution, host, i,  conv_mat_length);


    for( i = 0 ; i < info.nb_streams ; ++i ) hipStreamCreate( &streams[ i ] );

    GpuImgTransformStream::initStreamAndDevMem(per_stream_info, info.nb_streams, streams, dev_rgbs, host.rgb.in);

    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Mesure du temps de calcul du kernel uniquement.
    hipEventRecord(start);

    dim3 grid0((cols - 1) / (info.block.x - 1 + info.conv_properties.start_index) + 1,
               (rows - 1) / (info.block.y - 1 + info.conv_properties.start_index) + 1);


    for( i = 0 ; i < info.nb_streams ; ++i )
    {
        transform_img_stream_shared<<< grid0, info.block, 3 * info.block.x * info.block.y, streams[i]>>>(
                dev_rgbs[i].in, dev_rgbs[i].out,
                cols, per_stream_info.rows , dev_convolution.matrix, dev_convolution.prop);
    }


    for( int kth_pass = 1; kth_pass < info.nb_pass; kth_pass++){
        GpuImgTransformStream::swapStreamMem(per_stream_info, info.nb_streams, dev_rgbs);
        for( i = 0 ; i < info.nb_streams ; ++i )
        {
            transform_img_stream_shared<<< grid0, info.block, 3 * info.block.x * info.block.y, streams[i]>>>(
                    dev_rgbs[i].in, dev_rgbs[i].out,
                    cols, per_stream_info.rows , dev_convolution.matrix, dev_convolution.prop);
        }
    }

    hipDeviceSynchronize();
    hipEventRecord(stop);

    for(  i = 0 ; i < info.nb_streams ; ++i )
    {
        unsigned char *desti = m_out.data + (long)(i * (long)per_stream_info.size_effective);

        hipMemcpyAsync( desti, dev_rgbs[i].out,(long)per_stream_info.size_effective * sizeof(unsigned char),
                         hipMemcpyDeviceToHost, streams[ i ] );
    }

    hipEventSynchronize(stop);

    for( i = 0 ; i < info.nb_streams ; ++i ) hipStreamDestroy( streams[ i ] );

    float duration;
    hipEventElapsedTime(&duration, start, stop);
    std::cout << "time=" << duration << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    GpuImgTransformStream::freeMemory(dev_rgbs, dev_convolution, host, info.nb_streams);
    return 0;
}

