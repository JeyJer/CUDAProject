#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>
#include <cstring>

//----------------
//--- UTILITY ----
//----------------

//-- PARAMETERS --

void printParameters( std::string txtBold, std::string txtNormal, bool isTxtBoldUnderlined )
{
    std::cout << "\033[1" << ((isTxtBoldUnderlined) ? ";4" : "") << "m" << txtBold << "\033[0m" << txtNormal << std::endl;
}

void initParameters( std::string * img_in_path, std::string * img_out_path, bool * useShared,
    std::vector<std::string> * filtersEnabled, std::vector<int> * passNumber,
    int argc , char **argv )
{
    std::cout << std::endl;

    // Retrieve program parameters
    *img_in_path = argv[1];
    *img_out_path = argv[2];

    // Save if the program will use shared memory.
    *useShared = std::atoi( argv[3] );

    for( int i = 4 ; i < argc ; i+=2 )
    {
        filtersEnabled->push_back( argv[i] );
        passNumber->push_back( std::atoi(argv[i+1]) );
    }

    printParameters( "• Files path :", "", false );
    printParameters( "In path :", " "+(*img_in_path), true );
    printParameters( "Out path :", " "+(*img_out_path), true );
    std::cout << std::endl;

    printParameters( "• CUDA Options :", "", false );
    printParameters( "Memory Shared enabled ?", ((*useShared) ? " Yes" : " No"), true );
    std::cout << std::endl;

    printParameters( "• Image filters :", "", false );
    for( int i = 0 ; i < filtersEnabled->size() ; ++i )
    {
        printParameters( filtersEnabled->at(i) + " :", " "+std::to_string(passNumber->at(i)) + "-pass.", true );
    }
    std::cout << std::endl;
}

void presavedParameters( std::string* img_in_path, std::string* img_out_path, bool* useShared,
    std::vector<std::string>* filtersEnabled, std::vector<int> * passNumber )
{
    *img_in_path = "./in.jpg";
    *img_out_path = "./out.jpg";
    *useShared = 0;
    filtersEnabled->push_back( "BoxBlur" );
    passNumber->push_back( 10 );
}

//---- FILTERS ---

int init_divider( std::string filter )
{
    if( filter.compare("boxblur") )
    {
        return 9;
    }
    else if( filter.compare("gaussianblur") )
    {
        return 16;
    }
    else
    {
        return 1;
    }
}

char ** init_edge_detection_matrix()
{
    char ** conv_matrix = new char*[ 3 ];
    for( int i = 0; i < 3; ++i )
        conv_matrix[ i ]  = new char[ 3 ];

    conv_matrix[0][0] = -1;
    conv_matrix[0][1] = -1;
    conv_matrix[0][2] = -1;
    conv_matrix[1][0] = -1;
    conv_matrix[1][1] = 8;
    conv_matrix[1][2] = -1;
    conv_matrix[2][0] = -1;
    conv_matrix[2][1] = -1;
    conv_matrix[2][2] = -1;

    return conv_matrix;
}

char ** init_sharpen_matrix()
{
    char ** conv_matrix = new char*[ 3 ];
    for( int i = 0; i < 3; ++i )
        conv_matrix[ i ]  = new char[ 3 ];

    conv_matrix[0][0] = 0;
    conv_matrix[0][1] = -1;
    conv_matrix[0][2] = 0;
    conv_matrix[1][0] = -1;
    conv_matrix[1][1] = 5;
    conv_matrix[1][2] = -1;
    conv_matrix[2][0] = 0;
    conv_matrix[2][1] = -1;
    conv_matrix[2][2] = 0;

    return conv_matrix;
}

char ** init_box_blur_matrix()
{
    char ** conv_matrix = new char*[ 3 ];
    for( int i = 0; i < 3; ++i )
        conv_matrix[ i ]  = new char[ 3 ];

    conv_matrix[0][0] = 1;
    conv_matrix[0][1] = 1;
    conv_matrix[0][2] = 1;
    conv_matrix[1][0] = 1;
    conv_matrix[1][1] = 1;
    conv_matrix[1][2] = 1;
    conv_matrix[2][0] = 1;
    conv_matrix[2][1] = 1;
    conv_matrix[2][2] = 1;

    return conv_matrix;
}

char ** init_gaussian_blur_matrix()
{
    char ** conv_matrix = new char*[ 3 ];
    for( int i = 0; i < 3; ++i )
        conv_matrix[ i ]  = new char[ 3 ];

    conv_matrix[0][0] = 1;
    conv_matrix[0][1] = 2;
    conv_matrix[0][2] = 1;
    conv_matrix[1][0] = 2;
    conv_matrix[1][1] = 4;
    conv_matrix[1][2] = 2;
    conv_matrix[2][0] = 1;
    conv_matrix[2][1] = 2;
    conv_matrix[2][2] = 1;

    return conv_matrix;
}

char ** init_conv_matrix( std::string filter )
{
    if( filter.compare("edgedetection") == 0 )
    {
        return init_edge_detection_matrix();
    }
    else if( filter.compare("sharpen") == 0 )
    {
        return init_sharpen_matrix();
    }
    else if( filter.compare("boxblur") == 0 )
    {
        return init_box_blur_matrix();
    }
    else if( filter.compare("gaussianblur") == 0 )
    {
        return init_gaussian_blur_matrix();
    }
    else
    {
        std::cout << "The filter " << filter << " is unknowned." << std::endl;
        return nullptr;
    }
}

//---- POINTER MANIPULATION ----

void invert_pointer( unsigned char * ptr1, unsigned char * ptr2 )
{
    unsigned char* invertion_ptr = ptr1;
    ptr1 = ptr2;
    ptr2 = invertion_ptr;
}

void free_conv_matrix( char ** array )
{
    for( int i = 0 ; i < 3 ; i++ )
        delete[] array[i];
    delete[] array;
}

//----------------
//----- CUDA -----
//----------------

//---- CHRONO ----

void initCudaChrono( hipEvent_t * start, hipEvent_t * stop )
{
    hipEventCreate( start );
    hipEventCreate( stop );
}

void recordCudaChrono( hipEvent_t * chrono )
{
    hipEventRecord( *chrono );
}

float getCudaChronoTimeElapsed( hipEvent_t * start, hipEvent_t * stop )
{
    float duration;
    hipEventElapsedTime( &duration, *start, *stop );
    return duration;
}

void destroyCudaChrono( hipEvent_t * start, hipEvent_t * stop )
{
    hipEventDestroy( *start );
    hipEventDestroy( *stop );
}

//---- PROCESSING ----

__global__ void image_processing(unsigned char* rgb, unsigned char* s, std::size_t cols, std::size_t rows, char ** matrix, int divider )
{

    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    auto j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i > 0 && i < cols && j > 0 && j < rows)
    {
        auto h_r = matrix[0][0] * rgb[3 * ((j - 1) * cols + i - 1)] + matrix[0][1] * rgb[3 * ((j - 1) * cols + i)] + matrix[0][2] * rgb[3 * ((j - 1) * cols + i + 1)]
                 + matrix[1][0] * rgb[3 * ((j    ) * cols + i - 1)] + matrix[1][1] * rgb[3 * ((j    ) * cols + i)] + matrix[1][2] * rgb[3 * ((j    ) * cols + i + 1)]
                 + matrix[2][0] * rgb[3 * ((j + 1) * cols + i - 1)] + matrix[2][1] * rgb[3 * ((j + 1) * cols + i)] + matrix[2][2] * rgb[3 * ((j + 1) * cols + i + 1)];

        auto h_g = matrix[0][0] * rgb[3 * ((j - 1) * cols + i - 1) + 1] + matrix[0][1] * rgb[3 * ((j - 1) * cols + i) + 1] + matrix[0][2] * rgb[3 * ((j - 1) * cols + i + 1) + 1]
                 + matrix[1][0] * rgb[3 * ((j    ) * cols + i - 1) + 1] + matrix[1][1] * rgb[3 * ((j    ) * cols + i) + 1] + matrix[1][2] * rgb[3 * ((j    ) * cols + i + 1) + 1]
                 + matrix[2][0] * rgb[3 * ((j + 1) * cols + i - 1) + 1] + matrix[2][1] * rgb[3 * ((j + 1) * cols + i) + 1] + matrix[2][2] * rgb[3 * ((j + 1) * cols + i + 1) + 1];

        auto h_b = matrix[0][0] * rgb[3 * ((j - 1) * cols + i - 1) + 2] + matrix[0][1] * rgb[3 * ((j - 1) * cols + i) + 2] + matrix[0][2] * rgb[3 * ((j - 1) * cols + i + 1) + 2]
                 + matrix[1][0] * rgb[3 * ((j    ) * cols + i - 1) + 2] + matrix[1][1] * rgb[3 * ((j    ) * cols + i) + 2] + matrix[1][2] * rgb[3 * ((j    ) * cols + i + 1) + 2]
                 + matrix[2][0] * rgb[3 * ((j + 1) * cols + i - 1) + 2] + matrix[2][1] * rgb[3 * ((j + 1) * cols + i) + 2] + matrix[2][2] * rgb[3 * ((j + 1) * cols + i + 1) + 2];

        s[3 * (j * cols + i)    ] = (h_r / divider);
        s[3 * (j * cols + i) + 1] = (h_g / divider);
        s[3 * (j * cols + i) + 2] = (h_b / divider);
    }
}

__global__ void image_processing_shared(unsigned char* rgb, unsigned char* s, std::size_t cols, std::size_t rows, char ** matrix, int divider)
{
    auto i_global = blockIdx.x * (blockDim.x - 2) + threadIdx.x;
    auto j_global = blockIdx.y * (blockDim.y - 2) + threadIdx.y;

    auto i = threadIdx.x;
    auto j = threadIdx.y;

    auto w = blockDim.x;
    auto height = blockDim.y;

    extern __shared__ unsigned char sh[];

    if (i_global < cols && j_global < rows)
    {
        sh[3 * (j * w + i)    ] = rgb[3 * (j_global * cols + i_global)    ];
        sh[3 * (j * w + i) + 1] = rgb[3 * (j_global * cols + i_global) + 1];
        sh[3 * (j * w + i) + 2] = rgb[3* ( j_global * cols + i_global) + 2];
    }

    __syncthreads();

    if (i_global < cols - 1 && j_global < rows - 1 && i > 0 && i < (w - 1) && j > 0 && j < (height - 1))
    {
        auto h_r = matrix[0][0] * sh[3 * ((j - 1) * w + i - 1)] + matrix[0][1] * sh[3 * ((j - 1) * w + i)] + matrix[0][2] * sh[3 * ((j - 1) * w + i + 1)]
                 + matrix[1][0] * sh[3 * ((j    ) * w + i - 1)] + matrix[1][1] * sh[3 * ((j    ) * w + i)] + matrix[1][2] * sh[3 * ((j    ) * w + i + 1)]
                 + matrix[2][0] * sh[3 * ((j + 1) * w + i - 1)] + matrix[2][1] * sh[3 * ((j + 1) * w + i)] + matrix[2][2] * sh[3 * ((j + 1) * w + i + 1)];

        auto h_g = matrix[0][0] * sh[3 * ((j - 1) * w + i - 1) + 1] + matrix[0][1] * sh[3 * ((j - 1) * w + i) + 1] + matrix[0][2] * sh[3 * ((j - 1) * w + i + 1) + 1]
                 + matrix[1][0] * sh[3 * ((j    ) * w + i - 1) + 1] + matrix[1][1] * sh[3 * ((j    ) * w + i) + 1] + matrix[1][2] * sh[3 * ((j    ) * w + i + 1) + 1]
                 + matrix[2][0] * sh[3 * ((j + 1) * w + i - 1) + 1] + matrix[2][1] * sh[3 * ((j + 1) * w + i) + 1] + matrix[2][2] * sh[3 * ((j + 1) * w + i + 1) + 1];

        auto h_b = matrix[0][0] * sh[3 * ((j - 1) * w + i - 1) + 2] + matrix[0][1] * sh[3 * ((j - 1) * w + i) + 2] + matrix[0][2] * sh[3 * ((j - 1) * w + i + 1) + 2]
                 + matrix[1][0] * sh[3 * ((j    ) * w + i - 1) + 2] + matrix[1][1] * sh[3 * ((j    ) * w + i) + 2] + matrix[1][2] * sh[3 * ((j    ) * w + i + 1) + 2]
                 + matrix[2][0] * sh[3 * ((j + 1) * w + i - 1) + 2] + matrix[2][1] * sh[3 * ((j + 1) * w + i) + 2] + matrix[2][2] * sh[3 * ((j + 1) * w + i + 1) + 2];

        s[3 * (j_global * cols + i_global)    ] = (h_r / divider);
        s[3 * (j_global * cols + i_global) + 1] = (h_g / divider);
        s[3 * (j_global * cols + i_global) + 2] = (h_b / divider);
    }
}

//----------------
//----- MAIN -----
//----------------

int main( int argc , char **argv )
{
    //---- Declarate and allocate parameters
    std::string *img_in_path = new std::string();
    std::string *img_out_path = new std::string();
    bool *useShared = new bool;
    std::vector<std::string> *filtersEnabled = new std::vector<std::string>();
    std::vector<int> *passNumber = new std::vector<int>();

    //---- Initialize parameters
    // RELEASE_MODE
    initParameters( img_in_path, img_out_path, useShared, filtersEnabled, passNumber, argc, argv );
    // DEBUG_MODE
    // presavedParameters( img_in_path, img_out_path, useShared, filtersEnabled, passNumber );

    //---- Retrieve image properties
    cv::Mat img_in_matrix = cv::imread( *img_in_path, cv::IMREAD_UNCHANGED );
    auto rows = img_in_matrix.rows;
    auto cols = img_in_matrix.cols;
    std::cout << "Rows ? " << rows << std::endl;
    std::cout << "Cols ? " << cols << std::endl;

    //---- Allocate a cv::Mat (host-side) to store the device result
    std::cout << "[BEFORE_PROCESSING] " << "Allocation" << std::endl;
    unsigned char* img_out_h = nullptr;
    hipHostMalloc( &img_out_h, 3 * rows * cols );
    cv::Mat img_out_matrix( rows, cols, CV_8UC3, img_out_h );

    //---- allocate and initialize image's pixel array (host-side)
    unsigned char* rgb = nullptr;
    hipHostMalloc( &rgb, 3 * rows * cols );
    std::memcpy( rgb, img_in_matrix.data, 3 * rows * cols );

    //---- allocate and initialize image's pixel array (device-side)
    unsigned char* rgb_d;
    unsigned char* result_d;
    hipMalloc( &rgb_d, 3 * rows * cols );
    hipMalloc( &result_d, 3 * rows * cols );
    hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );
    std::cout << "rgb_d[0] = " << (int)rgb_d[0] << std::endl;

    //---- Threads distribution
    // grid block
    dim3 block( 32, 4 );
    // grid for non-shared memory processing
    dim3 grid0( (cols - 1) / block.x + 1, (rows - 1) / block.y + 1 );
    // grid for shared memory processing
    dim3 grid1( (cols - 1) / (block.x - 2) + 1, (rows - 1) / (block.y - 2) + 1 );

    //---- Init and start chrono
    hipEvent_t start, stop;
    initCudaChrono( &start, &stop );

    //---- Launch image processing loop
    for( int i = 0 ; i < filtersEnabled->size() ; ++i )
    {
        // init the convolution matrix and the divider according to the filter
        std::cout << "[" << filtersEnabled->at(i) << "] " << "Init matrix" << std::endl;
        char ** conv_matrix = init_conv_matrix( filtersEnabled->at(i) );
        if( conv_matrix == nullptr ) continue;

        // Jusqu'ici c'est bon quoi !

        int divider = init_divider( filtersEnabled->at(i) );

        // apply the filter how many passes wished
        std::cout << "[" << filtersEnabled->at(i) << "] " << "Apply filters" << std::endl;
        for( int j = 0 ; j < passNumber->at(i) ; ++j )
        {
            recordCudaChrono( &start );
            if( !*useShared )
            {
                std::cout << "[" << filtersEnabled->at(i) << "] " << "Non-shared processing" << std::endl;
                image_processing<<< grid0, block >>>( rgb_d, result_d, cols, rows, conv_matrix, divider );
            }
            else
            {
                std::cout << "[" << filtersEnabled->at(i) << "] " << "Shared processing" << std::endl;
                image_processing_shared<<< grid1, block, 3 * block.x * block.y >>>( rgb_d, result_d, cols, rows, conv_matrix, divider );
            }
            //---- get chrono time elapsed
            std::cout << "[" << filtersEnabled->at(i) << "] " << "Stop chrono" << std::endl;
            recordCudaChrono( &stop );
            hipEventSynchronize( stop );
            float duration = getCudaChronoTimeElapsed( &start, &stop );
            std::cout << "Pass duration : " << duration << "ms" << std::endl;
            // TODO Do something with duration

            // invert rgb_d with result_d, for any other pass
            std::cout << "[" << filtersEnabled->at(i) << "] " << " Invert pointers" << std::endl;
            invert_pointer( rgb_d, result_d );
        }
        std::cout << "[" << filtersEnabled->at(i) << "] " << "Free matrix" << std::endl;
        free_conv_matrix( conv_matrix );
    }
    // cancel the rgb_d and result_d invertion, to put back the result in result_d
    invert_pointer( rgb_d, result_d );

    //---- Copy the result to cv::Mat
    std::cout << "[AFTER_PROCESSING] " << "Memcpy" << std::endl;
    std::cout << "result_d[0] = " << (int)result_d[0] << std::endl;
    hipMemcpy( img_out_h, result_d, 3 * rows * cols, hipMemcpyDeviceToHost );

    //---- Write img_out onto the disk
    std::cout << "OUT PATH : " << cv::String(*img_out_path) << std::endl;
    cv::imwrite( cv::String(*img_out_path), img_out_matrix );

    //---- Free memory
    // host-side
    std::cout << "[AFTER_PROCESSING] " << "Free" << std::endl;
    hipFree( rgb_d );
    hipFree( result_d );
    // device-side
    hipHostFree( img_out_h );
    hipHostFree( rgb );
    destroyCudaChrono( &start, &stop );

    return 0;
}
